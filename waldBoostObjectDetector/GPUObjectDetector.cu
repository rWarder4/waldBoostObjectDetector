#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "npp.h"
#include<hiprand.h>
#include<hiprand/hiprand_kernel.h>

#include "WBSSettings.h"

#include <stdio.h>

hipError_t wbsOnCuda(double*, int*, int, int, double*, int, int *, int);

__global__ void applyClassifiersKernel(double *imageData,  int *imageDataDescriptors, double *weakClassProv, int descriptorStep, int imageSize, int* numberOfThreadsInImage, int weakClassNum)
{
    int threadNumber = threadIdx.x;
	int blockNumber = blockIdx.x;
	int gridNumber = gridDim.x;

	int realThreadNumber = threadNumber + blockNumber*WARP_SIZE;

	int imageNumber = 0;
	int threadNumberOnThisImage = 0;

	// find thread number on this image
	int previousNOT = 0;
	for (int i = 0; i < descriptorStep; i++) {		
		int currentNOT = numberOfThreadsInImage[i];
		if (realThreadNumber >= previousNOT && realThreadNumber < currentNOT) {
			// we are in right image
			imageNumber = i;
			threadNumberOnThisImage = realThreadNumber - previousNOT;
			break;
		}
		previousNOT = currentNOT;
	}
	
	// find out which data we will need
	/*int width;
	int height;
	int dataDescriptorNumber;
	int threadNumberOnThisImage;
	for (int i = 0; i < descriptorStep; i++) {
		int tN = 0;
		int nexttN = 0;
		int w = imageDataDescriptors[i + descriptorStep];
		int h = 0;
		if (i + 1 >= descriptorStep) {
			height = (imageSize - imageDataDescriptors[i]) / width;
		}
		else {
			height = (imageDataDescriptors[i + 1] - imageDataDescriptors[i]) / width;
		}
		nexttN += ((width - SLIDING_WINDOW_SIZE) / SLIDING_WINDOW_STEP) * ((height - SLIDING_WINDOW_SIZE) / SLIDING_WINDOW_STEP);
		// if number of threads which need to work is bigger than the thread number among all threads, we start calculating with this data
		if (nexttN > realThreadNumber) {
			dataDescriptorNumber = i;
			width = w;
			height = h;
			threadNumberOnThisImage = realThreadNumber - tN;
			break;
		}
		tN = nexttN;
	}*/

	// get descriptors of data
	int dataStart = imageDataDescriptors[imageNumber];
	int width = imageDataDescriptors[imageNumber + descriptorStep];
	int realWidthDiff = imageDataDescriptors[imageNumber + 2*descriptorStep];
	int dataEnd = imageDataDescriptors[imageNumber + 3 * descriptorStep];

	// find out on which position the sliding window should be
	int windowNumber = threadNumberOnThisImage * SLIDING_WINDOW_STEP;
	int windowsOnLine = (width - SLIDING_WINDOW_SIZE) - SLIDING_WINDOW_STEP;

	windowNumber = windowNumber * SLIDING_WINDOW_STEP;

	int lineNumber = windowNumber / windowsOnLine;
	int columnNumber = windowNumber % windowsOnLine;

	// drop the area with some chance
	if (weakClassProv[threadNumberOnThisImage%weakClassNum] < 0.99) {
		return;
	}

	//imageData[dataStart+columnNumber + width*lineNumber] = 255.0;

	// this part was able to go through classifier, draw rectangel - set boundary to 255
	int rowNumber = lineNumber;
	int colNumber = columnNumber;
	for (int j = 0; j < SLIDING_WINDOW_SIZE; j++) {
		// draw horizontal lines
		imageData[dataStart + colNumber + width*rowNumber+j] = 255.0;
		imageData[dataStart + colNumber + width*rowNumber + j + SLIDING_WINDOW_SIZE*width] = 255.0;

		// draw vertical lines
		imageData[dataStart + colNumber + width*rowNumber + j*width] = 255.0;
		imageData[dataStart + colNumber + width*rowNumber + j*width+SLIDING_WINDOW_SIZE] = 255.0;
	}
}

int GPUObjectDetector(double* imageData, int* imageDataDescriptor, int imageDataDescriptorStep, int imageDataSize, double* weakClassProb, int weakClassNum)
{
	// create array which will deternime how much threads should work on which image from pyramid
	int *threadsOnImage = new int[imageDataDescriptorStep];
	int finalNumberOfThreads = 0;
	for (int i = 0; i < imageDataDescriptorStep; i++) {
		//fprintf(stdout, "Desciptor value: %d, %d, %d, %d\n", descriptors[i], descriptors[i + descriptorStep], descriptors[i + 2 * descriptorStep], descriptors[i + 3 * descriptorStep]);
		int width = imageDataDescriptor[i + imageDataDescriptorStep];
		int height = 0;
		if (i + 1 >= imageDataDescriptorStep) {
			height = (imageDataSize - imageDataDescriptor[i]) / width;
		}
		else {
			height = (imageDataDescriptor[i + 1] - imageDataDescriptor[i]) / width;
		}
		int threadsOnThisImage = ((width - SLIDING_WINDOW_SIZE) / SLIDING_WINDOW_STEP) * ((height - SLIDING_WINDOW_SIZE) / SLIDING_WINDOW_STEP);
		finalNumberOfThreads += threadsOnThisImage;
		threadsOnImage[i] = finalNumberOfThreads;
	}

    hipError_t cudaStatus = wbsOnCuda(imageData, imageDataDescriptor, imageDataDescriptorStep, imageDataSize, weakClassProb, weakClassNum, threadsOnImage, finalNumberOfThreads);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to detect object using waldboost.
hipError_t wbsOnCuda(double *image, int *descriptors, int descriptorStep, int imageSize, double* weakClassProb, int weakClassnum, int *threadsOnImage, int numberOfThread)
{
	double *dev_image = 0;
	int *dev_descriptors = 0;
	double *dev_weakClassProb = 0;
	int *dev_numOfThreadInImage = 0;
    hipError_t cudaStatus;

	// number of cuda devices
	int cudaDeviceCount;
	cudaStatus = hipGetDeviceCount(&cudaDeviceCount);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	fprintf(stdout, "number of CUDA devices: %d\n", cudaDeviceCount);
	for (int i = 0; i < cudaDeviceCount; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		fprintf(stdout, "device_%d: %s\n", i, prop.name);
	}

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one input/output).
    cudaStatus = hipMalloc((void**)&dev_image, imageSize * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_descriptors, descriptorStep*4 * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_weakClassProb, weakClassnum * sizeof(double));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

	cudaStatus = hipMalloc((void**)&dev_numOfThreadInImage, descriptorStep * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_image, image, imageSize * sizeof(double), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_descriptors, descriptors, descriptorStep*4 * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	cudaStatus = hipMemcpy(dev_weakClassProb, weakClassProb, weakClassnum * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_numOfThreadInImage, threadsOnImage, descriptorStep * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// ------- DATA ON GPU ----------

	int warpNumber = numberOfThread / WARP_SIZE;

	fprintf(stdout, "Number of threads: %d, Number of warps: %d, Warp size: %d\n", numberOfThread, warpNumber, WARP_SIZE);

    // Launch a kernel on the GPU with one thread for each element.
    applyClassifiersKernel<<<warpNumber+WARP_SIZE-1, WARP_SIZE>>>(dev_image, dev_descriptors, dev_weakClassProb, descriptorStep, imageSize, dev_numOfThreadInImage, weakClassnum);

	// ------- COMPUTATION DONE --------

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(image, dev_image, imageSize * sizeof(double), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
	hipFree(dev_image);
    hipFree(dev_descriptors);
    hipFree(dev_weakClassProb);
	hipFree(dev_numOfThreadInImage);
    
    return cudaStatus;
}
